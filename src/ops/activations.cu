#include "hip/hip_runtime.h"
#pragma warning disable
#include "model.h"
#include "cuda_helper.h"

template <int DIM>
Tensor FFModel::sigmoid(std::string name, const Tensor& input, const int output_shape[])
{
  Activation<DIM> *activation = new Activation<DIM>(*this, name, "sigmoid", input, output_shape);
  layers.push_back(activation);
  return activation->output;
}
template <int DIM>
Tensor FFModel::relu(std::string name, const Tensor& input, const int output_shape[])
{
  Activation<DIM> *activation = new Activation<DIM>(*this, name, "relu", input, output_shape);
  layers.push_back(activation);
  return activation->output;
}
template <int DIM>
Tensor FFModel::elu(std::string name, const Tensor& input, const int output_shape[])
{
  Activation<DIM> *activation = new Activation<DIM>(*this, name, "elu", input, output_shape);
  layers.push_back(activation);
  return activation->output;
}
template <int DIM>
Tensor FFModel::identity(std::string name, const Tensor& input, const int output_shape[])
{
  Activation<DIM> *activation = new Activation<DIM>(*this, name, "identity", input, output_shape);
  layers.push_back(activation);
  return activation->output;
}
template <int DIM>
Tensor FFModel::tanh(std::string name, const Tensor& input, const int output_shape[])
{
  Activation<DIM> *activation = new Activation<DIM>(*this, name, "tanh", input, output_shape);
  layers.push_back(activation);
  return activation->output;
}



template <int DIM>
Activation<DIM>::Activation(FFModel& model,
  const std::string& pcname,
  const std::string& _mode,
  const Tensor& _input, 
  const int output_shape[])
  : Op(pcname, _input)
{
  task_is = IndexSpaceT<DIM>(model.get_or_create_task_is(DIM, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<DIM> part_rect = runtime->get_index_space_domain(ctx, task_is);
  // Create output tensor
  output = model.create_tensor<DIM>(output_shape, task_is, DT_FLOAT);
  model.create_data_parallel_partition_with_diff_dims<DIM, DIM>(
      _input, task_is, input_lps[0], input_grad_lps[0]);
  mode = _mode;
}

template <int DIM>
void Activation<DIM>::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<DIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<DIM> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    if (mode == "relu") {
      handle.mode = HIPDNN_ACTIVATION_RELU;
    } else if (mode == "sigmoid") {
      handle.mode = HIPDNN_ACTIVATION_SIGMOID;
    } else if (mode == "elu") {
      handle.mode = HIPDNN_ACTIVATION_ELU;
    } else if (mode == "tanh") {
      handle.mode = HIPDNN_ACTIVATION_TANH;
    } else if (mode == "idendity") {
      handle.mode = HIPDNN_ACTIVATION_PATHTRU;
    } else {
      throw 255;
    }
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  auto task_id = ACTIVATION_3D_INIT_TASK_ID;
  if (DIM == 3) {
    task_id = ACTIVATION_3D_INIT_TASK_ID;
  } else if (DIM == 2) {
    task_id = ACTIVATION_2D_INIT_TASK_ID;
  } else if (DIM == 1) {
    task_id = ACTIVATION_1D_INIT_TASK_ID;
  }
  else {
    printf("idim %d odim %d not supported", DIM, DIM);
  }
  IndexLauncher launcher(task_id, task_is,
    TaskArgument(this, sizeof(Activation)), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, inputs[0].region));



  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<DIM> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

template <int DIM>
OpMeta* Activation<DIM>::init_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  FFHandler handle = *((const FFHandler*) task->local_args);
  ActivationMeta* m = new ActivationMeta(handle);
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  TensorAccessorR<float, DIM> acc_input(regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, DIM> acc_output(regions[1], task->regions[1], FID_DATA, ctx, runtime);

#ifndef DISABLE_COMPUTATION
  // assert(rect_input == rect_output);
  int dims[DIM];
  int dims_buf[DIM];
  int stride[DIM];
  int stride_buf[DIM];
  stride_buf[0] = 1;
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->inputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&m->activation));
  checkCUDNN(hipdnnSetActivationDescriptor(
    m->activation,
    // HIPDNN_ACTIVATION_TANH,
    m->handle.mode,
    HIPDNN_NOT_PROPAGATE_NAN,
    0.0
  ));   
  if (DIM == 1) {
    int batch_size = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
      HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT,
      batch_size, 1, 1, 1));
  }
  else if (DIM == 2) {
    int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
    int batch_size = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
      HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT,
      batch_size, in_dim, 1, 1));
      // 1, 1, batch_size, in_dim));
  }
  else if (DIM > 2) {
    // cuda tensor dims order from outer to inner , so dims[0] is batch_dimension
    for (int i = 0; i < DIM; i++) {
      dims_buf[i] = acc_input.rect.hi[i] - acc_input.rect.lo[i] + 1;
      if (i + 1 < DIM) {
        stride_buf[i+1] = stride_buf[i] * dims_buf[i];
      }
    }
    for (int i = 0; i < DIM; i++) {
      dims[i] = dims_buf[DIM-i-1];
      stride[i] = stride_buf[DIM-i-1];
    }
    /*
    https://docs.nvidia.com/deeplearning/sdk/cudnn-api/index.html#hipdnnSetTensorNdDescriptor
    Note: Do not use for 2 dimensional tensors. The minimum number of dimensions in the filter descriptor is three. For more information, see hipdnnGetRNNLinLayerBiasParams().
    */
    checkCUDNN(hipdnnSetTensorNdDescriptor(m->inputTensor,
                                          HIPDNN_DATA_FLOAT,
                                          DIM,
                                          dims,
                                          stride));
                                                 
  }
                              
#endif
  return m;
}





/*
  regions[0](I): input
  regions[1](O): output
*/  
template <int DIM>
void Activation<DIM>::forward_task(const Task *task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  float alpha = 1.0f, beta = 0.0f;
  const ActivationMeta* m = *((ActivationMeta**) task->local_args);
  TensorAccessorR<float, DIM> acc_input(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, DIM> acc_output(
    regions[1], task->regions[1], FID_DATA, ctx, runtime,
    false/*readOutput*/);
#ifndef DISABLE_LEGION_CUDA_HIJACK
    hipStream_t stream;
    checkCUDA(hipStreamCreate(&stream));
    checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  // DOUBLE CHECK HANDLE TO PREVENT SEGMENTATION FAULT
  checkCUDA(hipdnnActivationForward(
    m->handle.dnn,
    m->activation,
    &alpha, m->inputTensor, acc_input.ptr,
    &beta, m->inputTensor, acc_output.ptr
  ));

}

template <int DIM>
void Activation<DIM>::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<DIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<DIM> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  auto task_id = ACTIVATION_3D_FWD_TASK_ID;
  if (DIM == 3) {
    task_id = ACTIVATION_3D_FWD_TASK_ID;
  } else if (DIM == 2) {
    task_id = ACTIVATION_2D_FWD_TASK_ID;
  } else if (DIM == 1) {
    task_id = ACTIVATION_1D_FWD_TASK_ID;
  } else {
    printf("idim %d odim %d not supported", DIM, DIM);
  }
  IndexLauncher launcher(task_id, task_is,
    TaskArgument(NULL, 0), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I) : input
  regions[1](I) : output
  regions[2](O) : input_grad
  regions[3](I) : output_grad
*/
template <int DIM>
void Activation<DIM>::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const ActivationMeta* m = *((ActivationMeta**) task->local_args);
  TensorAccessorR<float, DIM> acc_input(
    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, DIM> acc_output(
    regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, DIM> acc_output_grad(
    regions[3], task->regions[3], FID_DATA, ctx, runtime);
  TensorAccessorW<float, DIM> acc_input_grad(
    regions[2], task->regions[2], FID_DATA, ctx, runtime,
    false/*readOutput*/);


#ifndef DISABLE_LEGION_CUDA_HIJACK
    hipStream_t stream;
    checkCUDA(hipStreamCreate(&stream));
    checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif

  checkCUDA(hipdnnActivationBackward(
    m->handle.dnn,
    m->activation,
    &alpha, 
    m->inputTensor, acc_output.ptr,
    m->inputTensor, acc_output_grad.ptr,
    m->inputTensor, acc_input.ptr,
    &beta, m->inputTensor, acc_input_grad.ptr
  ));
}

template <int DIM>
void Activation<DIM>::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<DIM> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<DIM> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  auto task_id = ACTIVATION_3D_BWD_TASK_ID;
  if (DIM == 3) {
    task_id = ACTIVATION_3D_BWD_TASK_ID;
  } else if (DIM == 2) {
    task_id = ACTIVATION_2D_BWD_TASK_ID;
  } else if (DIM == 1) {
    task_id = ACTIVATION_1D_BWD_TASK_ID;
  } else {
    printf("idim %d odim %d not supported", DIM, DIM);
  }
  IndexLauncher launcher(task_id, task_is,
    TaskArgument(NULL, 0), argmap,
    Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
    FFConfig::get_hash_id(std::string(name)));
  
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(input_grad_lps[0], 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection id*/,
        READ_ONLY, EXCLUSIVE, output.region_grad));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}


template Activation<1>::Activation(FFModel& model,
  const std::string& pcname,
  const std::string& _mode,
  const Tensor& _input,
  const int output_shape[]);
template Activation<2>::Activation(FFModel& model,
  const std::string& pcname,
  const std::string& _mode,
  const Tensor& _input,
  const int output_shape[]);
template Activation<3>::Activation(FFModel& model,
  const std::string& pcname,
  const std::string& _mode,
  const Tensor& _input,
  const int output_shape[]);
template OpMeta* Activation<1>::init_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);  
template OpMeta* Activation<2>::init_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template OpMeta* Activation<3>::init_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<1>::init(const FFModel& ff);
template void Activation<2>::init(const FFModel& ff);
template void Activation<3>::init(const FFModel& ff);
template void Activation<1>::forward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<2>::forward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<3>::forward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<1>::forward(const FFModel& ff);
template void Activation<2>::forward(const FFModel& ff);
template void Activation<3>::forward(const FFModel& ff);
template void Activation<1>::backward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<2>::backward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<3>::backward_task(const Task *task,
  const std::vector<PhysicalRegion> &regions,
  Context ctx, Runtime *runtime);
template void Activation<1>::backward(const FFModel& ff);
template void Activation<2>::backward(const FFModel& ff);
template void Activation<3>::backward(const FFModel& ff);

template Tensor FFModel::relu<3>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::relu<2>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::relu<1>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::elu<3>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::elu<2>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::elu<1>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::sigmoid<3>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::sigmoid<2>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::sigmoid<1>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::identity<3>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::identity<2>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::identity<1>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::tanh<3>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::tanh<2>(std::string name, const Tensor& input, const int output_shape[]);
template Tensor FFModel::tanh<1>(std::string name, const Tensor& input, const int output_shape[]);
