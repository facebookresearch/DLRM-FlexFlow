#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::embedding(const std::string& pcname,
                          const Tensor& input,
                          int num_entries,
                          int out_dim,
                          AggrMode aggr,
                          Initializer* kernel_initializer)
{
  //assert(config.strategies.find(name) != config.strategies.end());
  //ParallelConfig pc = config.strategies[name];
  //IndexSpaceT<2> task_is = IndexSpaceT<2>(get_or_create_task_is(pc));
  Embedding* embed = new Embedding(*this, pcname, input, num_entries,
                                   out_dim, aggr, kernel_initializer);
  layers.push_back(embed);
  Parameter kernel;
  kernel.tensor = embed->kernel;
  kernel.op = embed;
  parameters.push_back(kernel);
  return embed->output;
}

Embedding::Embedding(FFModel& model,
                     const std::string& pcname,
                     const Tensor& _input,
                     //std::stirng name,
                     int num_entries, int outDim,
                     AggrMode _aggr,
                     Initializer* kernel_initializer)
: Op(pcname, _input), aggr(_aggr), profiling(model.config.profiling)
{
  assert(_input.numDim == 2);
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(pcname));
  
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  // Currently assume we can only partition over the sample dim
  assert(part_rect.hi[0] == part_rect.lo[0]);
  {
    const int dims[2] = {inputs[0].adim[1], outDim};
    output = model.create_tensor<2>(dims, task_is, DT_FLOAT);
  }
  {
    const int dims[2] = {outDim, num_entries};
    kernel = model.create_weight<2>(dims, task_is, DT_FLOAT, kernel_initializer);
  }
#ifdef DEADCODE
  // Create kernel tensor
  Rect<2> kernel_rect(Point<2>(0, 0), Point<2>(outDim-1, inDim-1));
  FieldSpace fs = runtime->create_field_space(ctx);
  FieldAllocator allocator = runtime->create_field_allocator(ctx, fs);
  allocator.allocate_field(sizeof(float), FID_DATA);
  IndexSpaceT<2> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  kernel.region = runtime->create_logical_region(ctx, kernel_is, fs);
  {
    int num_part_c = part_rect.hi[0] - part_rect.lo[0] + 1;
    int extent_c = (outDim + num_part_c - 1) / num_part_c;
    Rect<2> extent(Point<2>(0, 0), Point<2>(extent_c, inDim-1));
    Transform<2, 2> transform;
    transform[0][0] = extent_c; transform[0][1] = 0;
    transform[1][0] = 0; transform[1][1] = 0;
    IndexPartition ip = runtime->create_partition_by_restriction(
        ctx, kernel_is, task_is, transform, extent);
    kernel.part = runtime->get_logical_partition(
        ctx, kernel.region, ip);
  }
  // Create kernel tensor gradients
  Rect<3> kernel_grad_rect(Point<3>(0, 0, 0),
      Point<3>(outDim-1, inDim-1, part_rect.hi[1] - part_rect.lo[1]));
  IndexSpaceT<3> kernel_grad_is = runtime->create_index_space(
      ctx, kernel_grad_rect);
  kernel.region_grad = runtime->create_logical_region(
      ctx, kernel_grad_is, fs);
  {
    int num_part_c = part_rect.hi[0] - part_rect.lo[0] + 1;
    int extent_c = (outDim + num_part_c - 1) / num_part_c;
    Rect<3> extent(Point<3>(0, 0, 0), Point<3>(extent_c, inDim-1, 0));
    Transform<3, 2> transform;
    transform[0][0] = extent_c; transform[0][1] = 0;
    transform[1][0] = 0; transform[1][1] = 0;
    transform[2][0] = 0; transform[2][1] = 1;
    IndexPartition ip = runtime->create_partition_by_restriction(
        ctx, kernel_grad_is, task_is, transform, extent);
    kernel.part_grad = runtime->get_logical_partition(
        ctx, kernel.region_grad, ip);
    assert(runtime->is_index_partition_disjoint(ctx, ip));
    assert(runtime->is_index_partition_complete(ctx, ip));
  }
#endif
  // Compute partition bound for input
  Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    // Currently assert input must have the same partition
    // to avoid data movement
    assert(false);
  }
}

//__host__
//OpMeta* Embedding::init_task(const Task *task,
//                             const std::vector<PhysicalRegion> &regions,
//                             Context ctx, Runtime* runtime)
//{}

void Embedding::init(const FFModel& ff)
{}

__global__
void embed_forward(const int64_t* input,
                   float* output,
                   const float* embed,
                   int out_dim,
                   int in_dim,
                   int batch_size,
                   AggrMode aggr)
{
  CUDA_KERNEL_LOOP(i, batch_size * out_dim)
  {
    output[i] = 0;
    int idx = i / out_dim;
    int off = i % out_dim;
    for (int j = 0; j < in_dim; j++) {
      int64_t wordIdx = input[idx * in_dim + j];
      output[i] += embed[wordIdx * out_dim + off];
      if (aggr == AGGR_MODE_SUM) {
      } else {
        assert(aggr == AGGR_MODE_AVG);
        output[i] /= in_dim;
      }
    }
  }
}

__global__
void embed_backward(const int64_t* input,
                    const float* output,
                    float* embed,
                    int out_dim,
                    int in_dim,
                    int batch_size,
                    AggrMode aggr)
{
  CUDA_KERNEL_LOOP(i, batch_size * out_dim)
  {
    int idx = i / out_dim;
    int off = i % out_dim;
    float gradient;
    if (aggr == AGGR_MODE_SUM) {
       gradient = output[i];
    } else {
      assert(aggr == AGGR_MODE_AVG);
      gradient = output[i] / in_dim;
    }
    for (int j = 0; j < in_dim; j++) {
      int64_t wordIdx = input[idx * in_dim + j];
      atomicAdd(embed + wordIdx * out_dim + off, gradient);
    }
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I): kernel
*/
__host__
void Embedding::forward_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Embedding* embed = (Embedding*) task->args;
  TensorAccessorR<int64_t, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  TensorAccessorR<float, 2> accWeight(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  // Input matches Output
  assert(accInput.rect.hi[1] == accOutput.rect.hi[1]);
  assert(accInput.rect.lo[1] == accOutput.rect.lo[1]);
  // Weight matches Output
  assert(accWeight.rect.hi[1] == accOutput.rect.hi[0]);
  assert(accWeight.rect.lo[1] == accOutput.rect.lo[0]);
  int in_dim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  int out_dim = accOutput.rect.hi[0] - accOutput.rect.lo[0] + 1;
  int batch_size = accOutput.rect.hi[1] - accOutput.rect.lo[1] + 1;
  embed_forward<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput.ptr, accOutput.ptr, accWeight.ptr, out_dim, in_dim, batch_size, embed->aggr);
  checkCUDA(hipDeviceSynchronize());
  if (embed->profiling) {
    print_tensor<2, int64_t>(accInput.ptr, accInput.rect, "[Embedding:forward:input]");
    print_tensor<2, float>(accWeight.ptr, accWeight.rect, "[Embedding:forward:weight]");
    print_tensor<2, float>(accOutput.ptr, accOutput.rect, "[Embedding:forward:output]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Embedding::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(EMBED_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Embedding)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0]: input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1]: output
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection*/,
                        WRITE_ONLY, EXCLUSIVE, output.region,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(1, FID_DATA);
  // regions[2]: weight
  launcher.add_region_requirement(
      RegionRequirement(kernel.part, 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, kernel.region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

void Embedding::backward_task(const Task *task,
                              const std::vector<PhysicalRegion> &regions,
                              Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Embedding* embed = (Embedding*) task->args;
  TensorAccessorR<int64_t, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> accWeightGrad(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, false/*readOutput*/);
  // Input matches Output
  assert(accInput.rect.hi[1] == accOutput.rect.hi[1]);
  assert(accInput.rect.lo[1] == accOutput.rect.lo[1]);
  // WeightGrad matches Output
  assert(accWeightGrad.rect.hi[1] - accWeightGrad.rect.lo[1] == accOutput.rect.hi[0] - accOutput.rect.lo[0]);
  int in_dim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  int out_dim = accOutput.rect.hi[0] - accOutput.rect.lo[0] + 1;
  int batch_size = accOutput.rect.hi[1] - accOutput.rect.lo[1] + 1;
  // Explicitly initialize accWegihtGrad to zero to aviod calling zero_gradients() before backward()
  // as an optimization for DLRM
  assign_kernel<<<GET_BLOCKS(accWeightGrad.rect.volume()), CUDA_NUM_THREADS>>>(
        accWeightGrad.ptr, accWeightGrad.rect.volume(), 0.0f);
  embed_backward<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput.ptr, accOutput.ptr, accWeightGrad.ptr, out_dim, in_dim, batch_size, embed->aggr);
  checkCUDA(hipDeviceSynchronize());
  if (embed->profiling) {
    print_tensor<2, float>(accOutput.ptr, accOutput.rect, "[Embedding:backward:output_grad]");
    print_tensor<2, float>(accWeightGrad.ptr, accWeightGrad.rect, "[Embedding:backward:weight_grad]");
    print_tensor<2, int64_t>(accInput.ptr, accInput.rect, "[Embedding:backward:input]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Embedding::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(EMBED_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Embedding)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0]: input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1]: output_grad
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, output.region_grad,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(1, FID_DATA);
  // regions[2]: weight_grad
  launcher.add_region_requirement(
      RegionRequirement(kernel.part_grad, 0/*projection*/,
                        WRITE_ONLY, EXCLUSIVE, kernel.region_grad));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}
