#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::embedding(const Tensor& input,
                          int num_entries,
                          int out_dim,
                          AggrMode aggr,
                          const Op* shared_op,
                          Initializer* kernel_initializer)
{
  //assert(config.strategies.find(name) != config.strategies.end());
  //ParallelConfig pc = config.strategies[name];
  //IndexSpaceT<2> task_is = IndexSpaceT<2>(get_or_create_task_is(pc));
  Embedding* embed = new Embedding(*this, input, num_entries,
                                   out_dim, aggr, shared_op, kernel_initializer);
  layers.push_back(embed);
  return embed->outputs[0];
}

Embedding* FFModel::embedding(int num_entries,
                              int out_dim,
                              AggrMode aggr,
                              Initializer* kernel_initializer)
{
  //assert(config.strategies.find(name) != config.strategies.end());
  //ParallelConfig pc = config.strategies[name];
  //IndexSpaceT<2> task_is = IndexSpaceT<2>(get_or_create_task_is(pc));
  Embedding* embed = new Embedding(*this, num_entries,
                                   out_dim, aggr, kernel_initializer);
  layers.push_back(embed);
  return embed;
}

Embedding::Embedding(FFModel& model,
                     const Tensor& _input,
                     //std::stirng name,
                     int _num_entries, int outDim,
                     AggrMode _aggr,
                     const Op* shared_op,
                     Initializer* _kernel_initializer)
: Op(model, OP_EMBEDDING, shared_op, "Embed_"+std::to_string(_num_entries)+"x"+std::to_string(outDim), _input),
  num_entries(_num_entries), out_channels(outDim), aggr(_aggr),
  kernel_initializer(_kernel_initializer), profiling(model.config.profiling)
{
  assert(_input.numDim == 2);
  outputs[0].numDim = 2;
  outputs[0].adim[0] = out_channels;
  outputs[0].adim[1] = inputs[0].adim[1];
  weights[0].numDim = 2;
  weights[0].adim[0] = num_entries;
  weights[0].adim[1] = out_channels;
  numWeights = 1;
}

Embedding::Embedding(FFModel& model,
                     int _num_entries, int outDim,
                     AggrMode _aggr,
                     Initializer* kernel_initializer)
: Op(model, OP_EMBEDDING, "Embed_"+std::to_string(_num_entries)+"x"+std::to_string(outDim), 1),
  num_entries(_num_entries), out_channels(outDim), aggr(_aggr), profiling(model.config.profiling)
{
}

Tensor Embedding::init_inout(FFModel& model, const Tensor& _input)
{
  assert(_input.numDim == 2);
  inputs[0] = _input;
  create_output_and_partition(model);
  return outputs[0];
}

/*
void Embedding::add_to_model(FFModel& model)
{
  model.layers.push_back(this);
  model.parameters.push_back(weights[0]);
}
*/

void Embedding::create_weights(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));
  {
    const int dims[2] = {out_channels, num_entries};
    // Embeddding weights and linear weights can be partitioned in the same way
    weights[0] = model.create_linear_weight<2>(this, dims, (IndexSpaceT<2>)task_is, DT_FLOAT, kernel_initializer);
    assert(numWeights == 1);
  }
}

void Embedding::create_output_and_partition(FFModel& model)
{
  // Retrive the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(2, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  // Currently assume we can only partition over the sample dim
  assert(part_rect.hi[0] == part_rect.lo[0]);
  {
    const int dims[2] = {inputs[0].adim[1], out_channels};
    outputs[0] = model.create_tensor<2>(dims, DT_FLOAT, this);
    outputs[0].owner_op = this;
    outputs[0].owner_idx = 0;
  }
  // Compute partition bound for input
  Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    // Currently assert input must have the same partition
    // to avoid data movement
    assert(false);
  }
}

__host__
OpMeta* Embedding::init_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime* runtime)
{
  // We don't need EmbedMeta for forward or backward
  return NULL;
}

void Embedding::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(EMBED_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Embedding)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0]: input
  //launcher.add_region_requirement(
  //  RegionRequirement(input_lps[0], 0/*projection*/,
  //    READ_ONLY, EXCLUSIVE, inputs[0].region));
  //launcher.add_field(0, FID_DATA);
  // regions[1]: output
  launcher.add_region_requirement(
    RegionRequirement(outputs[0].part, 0/*projection*/,
      WRITE_ONLY, EXCLUSIVE, outputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[2]: weight
  launcher.add_region_requirement(
    RegionRequirement(weights[0].part, 0/*projection*/,
      READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(1, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__global__
void embed_forward(const int64_t* input,
                   float* output,
                   const float* embed,
                   int out_dim,
                   int in_dim,
                   int batch_size,
                   AggrMode aggr)
{
  CUDA_KERNEL_LOOP(i, batch_size * out_dim)
  {
    output[i] = 0;
    int idx = i / out_dim;
    int off = i % out_dim;
    for (int j = 0; j < in_dim; j++) {
      int64_t wordIdx = input[idx * in_dim + j];
      output[i] += embed[wordIdx * out_dim + off];
      if (aggr == AGGR_MODE_SUM) {
      } else {
        assert(aggr == AGGR_MODE_AVG);
        output[i] /= in_dim;
      }
    }
  }
}

__global__
void embed_backward(const int64_t* input,
                    const float* output,
                    float* embed,
                    int out_dim,
                    int in_dim,
                    int batch_size,
                    AggrMode aggr)
{
  CUDA_KERNEL_LOOP(i, batch_size * out_dim)
  {
    int idx = i / out_dim;
    int off = i % out_dim;
    float gradient;
    if (aggr == AGGR_MODE_SUM) {
       gradient = output[i];
    } else {
      assert(aggr == AGGR_MODE_AVG);
      gradient = output[i] / in_dim;
    }
    for (int j = 0; j < in_dim; j++) {
      int64_t wordIdx = input[idx * in_dim + j];
      atomicAdd(embed + wordIdx * out_dim + off, gradient);
    }
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I): kernel
*/
__host__
void Embedding::forward_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime* runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Embedding* embed = (Embedding*) task->args;
  TensorAccessorR<int64_t, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime, false/*readOutput*/);
  TensorAccessorR<float, 2> accWeight(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  // Input matches Output
  assert(accInput.rect.hi[1] == accOutput.rect.hi[1]);
  assert(accInput.rect.lo[1] == accOutput.rect.lo[1]);
  // Weight matches Output
  assert(accWeight.rect.hi[1] == accOutput.rect.hi[0]);
  assert(accWeight.rect.lo[1] == accOutput.rect.lo[0]);
  int in_dim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  int out_dim = accOutput.rect.hi[0] - accOutput.rect.lo[0] + 1;
  int batch_size = accOutput.rect.hi[1] - accOutput.rect.lo[1] + 1;
  embed_forward<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput.ptr, accOutput.ptr, accWeight.ptr, out_dim, in_dim, batch_size, embed->aggr);
  checkCUDA(hipDeviceSynchronize());
  if (embed->profiling) {
    print_tensor<2, int64_t>(accInput.ptr, accInput.rect, "[Embedding:forward:input]");
    print_tensor<2, float>(accWeight.ptr, accWeight.rect, "[Embedding:forward:weight]");
    print_tensor<2, float>(accOutput.ptr, accOutput.rect, "[Embedding:forward:output]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Embedding::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(EMBED_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Embedding)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0]: input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1]: output
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part, 0/*projection*/,
                        WRITE_ONLY, EXCLUSIVE, outputs[0].region,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(1, FID_DATA);
  // regions[2]: weight
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part, 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, weights[0].region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

void Embedding::backward_task(const Task *task,
                              const std::vector<PhysicalRegion> &regions,
                              Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Embedding* embed = (Embedding*) task->args;
  TensorAccessorR<int64_t, 2> accInput(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> accOutput(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> accWeightGrad(
      regions[2], task->regions[2], FID_DATA, ctx, runtime, true/*readOutput*/);
  // Input matches Output
  assert(accInput.rect.hi[1] == accOutput.rect.hi[1]);
  assert(accInput.rect.lo[1] == accOutput.rect.lo[1]);
  // WeightGrad matches Output
  assert(accWeightGrad.rect.hi[1] - accWeightGrad.rect.lo[1] == accOutput.rect.hi[0] - accOutput.rect.lo[0]);
  int in_dim = accInput.rect.hi[0] - accInput.rect.lo[0] + 1;
  int out_dim = accOutput.rect.hi[0] - accOutput.rect.lo[0] + 1;
  int batch_size = accOutput.rect.hi[1] - accOutput.rect.lo[1] + 1;
  // Explicitly initialize accWegihtGrad to zero to aviod calling zero_gradients() before backward()
  // as an optimization for DLRM
  //assign_kernel<<<GET_BLOCKS(accWeightGrad.rect.volume()), CUDA_NUM_THREADS>>>(
  //      accWeightGrad.ptr, accWeightGrad.rect.volume(), 0.0f);
  embed_backward<<<GET_BLOCKS(accOutput.rect.volume()), CUDA_NUM_THREADS>>>(
      accInput.ptr, accOutput.ptr, accWeightGrad.ptr, out_dim, in_dim, batch_size, embed->aggr);
  checkCUDA(hipDeviceSynchronize());
  if (embed->profiling) {
    print_tensor<2, float>(accOutput.ptr, accOutput.rect, "[Embedding:backward:output_grad]");
    print_tensor<2, float>(accWeightGrad.ptr, accWeightGrad.rect, "[Embedding:backward:weight_grad]");
    print_tensor<2, int64_t>(accInput.ptr, accInput.rect, "[Embedding:backward:input]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Embedding::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  IndexLauncher launcher(EMBED_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Embedding)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // regions[0]: input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1]: output_grad
  launcher.add_region_requirement(
      RegionRequirement(outputs[0].part_grad, 0/*projection*/,
                        READ_ONLY, EXCLUSIVE, outputs[0].region_grad,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(1, FID_DATA);
  // regions[2]: weight_grad
  launcher.add_region_requirement(
      RegionRequirement(weights[0].part_grad, 0/*projection*/,
                        READ_WRITE, EXCLUSIVE, weights[0].region_grad));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

bool Embedding::measure_compute_time(Simulator* sim,
                                     const ParallelConfig& pc,
                                     float& forward_time,
                                     float& backward_time)
{
  //TODO: implement measure_forward
  return false;
}
