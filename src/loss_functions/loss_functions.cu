#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Loss::Loss(const std::string& loss)
{
  if (loss == "categorical_crossentropy")
    loss_type = LOSS_CATEGORICAL_CROSSENTROPY;
  else if (loss == "sparse_categorical_crossentropy")
    loss_type = LOSS_SPARSE_CATEGORICAL_CROSSENTROPY;
  else if (loss == "mean_squared_error")
    loss_type = LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE;
  else
    // Unrecognized loss type
    assert(false);
}

Loss::Loss(LossType _loss_type)
: loss_type(_loss_type)
{}

__global__
void sparse_categorical_crossentropy_loss_backward(
    float *logit_grad,
    const int *label,
    coord_t num_samples,
    coord_t num_classes)
{
  CUDA_KERNEL_LOOP(i, num_samples)
  {
    int label_idx = label[i];
    logit_grad[i * num_classes + label_idx] -= 1.0f;
  }
}

__global__
void categorical_crossentropy_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__global__
void mean_squared_error_avg_loss_backward(
    float *logit_grad,
    const float *logit,
    const float *label,
    coord_t num_elements)
{
  CUDA_KERNEL_LOOP(i, num_elements)
  {
    logit_grad[i] = logit[i] - label[i];
  }
}

__host__
void Loss::backward_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Loss* loss = (Loss*) task->args;
  if (loss->loss_type == LOSS_SPARSE_CATEGORICAL_CROSSENTROPY) {
    //sparse_categorical_crossentropy has label of dim: (batch_size, 1)
    TensorAccessorW<float, 2> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, 2> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<int, 2> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    int num_samples = acc_logit.rect.hi[1] - acc_logit.rect.lo[1] + 1;
    int num_classes = acc_logit.rect.hi[0] - acc_logit.rect.lo[0] + 1;
    assert(acc_logit_grad.rect == acc_logit.rect);
    assert(acc_label.rect.hi[1] == acc_logit.rect.hi[1]);
    assert(acc_label.rect.lo[1] == acc_logit.rect.lo[1]);
    assert(acc_label.rect.lo[0] == acc_label.rect.hi[0]);
    checkCUDA(hipMemcpy(acc_logit_grad.ptr, acc_logit.ptr,
                         acc_logit.rect.volume() * sizeof(float),
                         hipMemcpyDeviceToDevice));
    sparse_categorical_crossentropy_loss_backward<<<GET_BLOCKS(num_samples), CUDA_NUM_THREADS>>>(
        acc_logit_grad.ptr, acc_label.ptr, num_samples, num_classes);
    // Scale logit gradients by op->scale_factor
    scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
  } else {
    TensorAccessorW<float, 2> acc_logit_grad(
        regions[0], task->regions[0], FID_DATA, ctx, runtime,
        true/*readOutput*/);
    TensorAccessorR<float, 2> acc_logit(
        regions[1], task->regions[1], FID_DATA, ctx, runtime);
    TensorAccessorR<float, 2> acc_label(
        regions[2], task->regions[2], FID_DATA, ctx, runtime);
    // other loss require label and logit have identical shape
    assert(acc_logit.rect == acc_label.rect);
    assert(acc_logit_grad.rect == acc_logit.rect);
    int num_samples = acc_logit.rect.hi[1] - acc_logit.rect.lo[1] + 1;
    int num_channels = acc_logit.rect.hi[0] - acc_logit.rect.lo[0] + 1;
    if (loss->loss_type == LOSS_CATEGORICAL_CROSSENTROPY) {
      categorical_crossentropy_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else if (loss->loss_type == LOSS_MEAN_SQUARED_ERROR_AVG_REDUCE) {
      mean_squared_error_avg_loss_backward<<<GET_BLOCKS(acc_logit.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit.ptr, acc_label.ptr,
          acc_logit.rect.volume());
      // Scale logit gradients by loss->scale_factor
      scale_kernel<<<GET_BLOCKS(acc_logit_grad.rect.volume()), CUDA_NUM_THREADS>>>(
          acc_logit_grad.ptr, acc_logit_grad.rect.volume(), 0, loss->scale_factor);
    } else {
      fprintf(stderr, "Unsupported loss --- report this error to the FlexFlow developers\n");
      assert(false);
    }
  }
}

void Loss::backward(FFModel* model,
                    const Tensor* logit,
                    const Tensor* label)
{
  // Compute scale factor for loss backpropagation
  scale_factor = 1.0f/ logit->adim[logit->numDim-1];
  //scale_factor = 1.0f;
  // Use the same parallel strategy as the owner of logit
  std::string pcname = logit->owner_op->name;
  IndexSpaceT<2> task_is = IndexSpaceT<2>(model->get_or_create_task_is(2, pcname));
  Context ctx = model->config.lg_ctx;
  Runtime* runtime = model->config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  Rect<2> logit_rect = runtime->get_index_partition_color_space(
      ctx, logit->part.get_index_partition());
  Rect<2> label_rect = runtime->get_index_partition_color_space(
      ctx, label->part.get_index_partition());
  if((logit_rect != part_rect) || (label_rect != part_rect)) {
    fprintf(stderr, "Encounter inconsistency in parallelizing loss computation");
    assert(false);
  }
  ArgumentMap argmap;
  IndexLauncher launcher(LOSS_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Loss)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(pcname));
  launcher.add_region_requirement(
      RegionRequirement(logit->part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, logit->region_grad));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(logit->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, logit->region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(label->part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, label->region));
  launcher.add_field(2, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

