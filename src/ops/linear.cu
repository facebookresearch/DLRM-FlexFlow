#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::dense(std::string name,
                      const Tensor& input,
                      int outDim, 
                      ActiMode activation,
                      bool use_bias, 
                      Initializer* kernel_initializer,
                      Initializer* bias_initializer)
{
  if (kernel_initializer == NULL) {
    int seed = std::rand();
    kernel_initializer = new GlorotUniform(seed);
  }
  if (bias_initializer == NULL) {
    bias_initializer = new ZeroInitializer();
  }
  Linear *li = new Linear(*this, name, input, outDim, activation, use_bias,
                          kernel_initializer, bias_initializer);
  layers.push_back(li);
  Parameter kernel, bias;
  kernel.tensor = li->kernel;
  kernel.op = li;
  bias.tensor = li->bias;
  bias.op = li;
  parameters.push_back(kernel);
  parameters.push_back(bias);
  return li->output;
}

// Deprecated API -- TO BE REMOVED
Tensor FFModel::linear(std::string name,
                       const Tensor& input,
                       int out_dim,
                       ActiMode activation,
                       bool use_bias,
                       Initializer* kernel_initializer,
                       Initializer* bias_initializer)
{
  return dense(name, input, out_dim, activation,
               kernel_initializer, bias_initializer);
}

Linear::Linear(FFModel& model,
               const std::string& pcname,
               const Tensor& _input,
               int out_dim,
               ActiMode _activation,
               bool use_bias,
               Initializer* kernel_initializer,
               Initializer* bias_initializer)
: Op(pcname, _input), activation(_activation),
  profiling(model.config.profiling)
{
  assert(_input.numDim == 2);
  // Retrive the task indexspace for the op
  task_is = IndexSpaceT<2>(model.get_or_create_task_is(pcname));

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<2> part_rect = runtime->get_index_space_domain(ctx, task_is);
  int num_par_c = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_n = part_rect.hi[1] - part_rect.lo[1] + 1;
  int in_dim = _input.adim[0];
  int batch_size = _input.adim[1];
  {
    const int dims[2] = {batch_size, out_dim}; // out_dim, batch_size
    output = model.create_tensor<2>(dims, task_is, DT_FLOAT);
  }
  // Create kernel tensor
  {
    const int dims[2] = {out_dim, in_dim}; // target shape k,m
    kernel = model.create_weight<2>(dims, task_is, DT_FLOAT, kernel_initializer);
  }
  // Create bias tensor
  if (use_bias) {
    const int dims[1] = {out_dim};
    bias = model.create_weight<1>(dims, task_is, DT_FLOAT, bias_initializer);
  }
  // Compute partition bound for input
  Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  // Create replica tensor
  if (num_par_c > 1) {
    const int dims[3] = {num_par_c, batch_size, in_dim};
    replica = model.create_replica<3>(dims, task_is, DT_FLOAT);
    {
      Rect<2> extent(Point<2>(0, 0), Point<2>(in_dim-1, batch_size/num_par_n-1));
      Transform<2, 2> transform;
      transform[0][0] = 0;
      transform[0][1] = 0;
      transform[1][0] = 0;
      transform[1][1] = batch_size/num_par_n;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
    }
    // Backward use the same ip as inputs[0]
    input_grad_lps[0] = inputs[0].part_grad;
    {
      IndexSpaceT<2> input_task_is = IndexSpaceT<2>(model.get_or_create_task_is(input_rect));
      const coord_t num_parts[2] = {input_rect.hi[0] - input_rect.lo[0] + 1,
                                    input_rect.hi[1] - input_rect.lo[1] + 1};
      Rect<3> extent(Point<3>(0, 0, 0),
          Point<3>(in_dim/num_parts[0]-1, batch_size/num_parts[1]-1, num_par_c-1));
      Transform<3, 2> transform;
      for (int i = 0; i < 3; i++)
        for (int j = 0; j < 2; j++)
          transform[i][j] = 0;
      transform[0][0] = in_dim / num_parts[0];
      transform[1][1] = batch_size / num_parts[1];
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, replica.region_grad.get_index_space(), input_task_is,
          transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      // Note we use replica.part to save how to partition the replica
      // to compute input_grad_lps
      replica.part = runtime->get_logical_partition(
          ctx, replica.region_grad, ip);
    }
  } 
  else {
    if (input_rect == part_rect) {
      input_lps[0] = inputs[0].part;
      input_grad_lps[0] = inputs[0].part_grad;
    } else {
      Rect<2> extent(Point<2>(0,0), Point<2>(in_dim-1,batch_size/num_par_n-1));
      Transform<2, 2> transform;
      transform[0][0] = 0;
      transform[0][1] = 0;
      transform[1][0] = 0;
      transform[1][1] = batch_size / num_par_n;
      IndexPartition ip = runtime->create_partition_by_restriction(
          ctx, inputs[0].region.get_index_space(), task_is, transform, extent);
      assert(runtime->is_index_partition_disjoint(ctx, ip));
      assert(runtime->is_index_partition_complete(ctx, ip));
      input_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region, ip);
      input_grad_lps[0] = runtime->get_logical_partition(
          ctx, inputs[0].region_grad, ip);
    }
  }
}

/*
  regions[0](O): output
  regions[1](I): kernel
  regions[2](I): bias
*/
OpMeta* Linear::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 3);
  assert(task->regions.size() == 3);
  const Linear* linear = (Linear*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  //TensorAccessorR<float, 2> acc_input(
  //    regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_output(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  //int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int in_dim = acc_kernel.rect.hi[0] - acc_kernel.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_output.rect.hi[1] - acc_output.rect.lo[1] + 1;
  printf("init linear (input): in_dim(%d) out_dim(%d) batch_size(%d)\n",
      in_dim, out_dim, batch_size);
  LinearMeta* m = new LinearMeta(handle);

  float* dram_one_ptr = (float *) malloc(sizeof(float) * batch_size);
  for (int i = 0; i < batch_size; i++)
    dram_one_ptr[i] = 1.0f;
  float* fb_one_ptr;
  checkCUDA(hipMalloc(&fb_one_ptr, sizeof(float) * batch_size));
  checkCUDA(hipMemcpy(fb_one_ptr, dram_one_ptr,
                       sizeof(float) * batch_size, hipMemcpyHostToDevice));
  m->one_ptr = (const float*) fb_one_ptr;
  if (linear->activation != AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (linear->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batch_size, out_dim, 1, 1));
  }
  return m;
}

void Linear::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher launcher(LINEAR_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  //launcher.add_region_requirement(
  //    RegionRequirement(input_lps[0], 0/*projection id*/,
  //                      READ_ONLY, EXCLUSIVE, inputs[0].region));
  //launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(kernel.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, kernel.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(bias.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, bias.region));
  launcher.add_field(2, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I); input
  regions[1](O): output
  regions[2](I): kernel
  regions[3](I): bias
*/
__host__
void Linear::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorR<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_output(
      regions[1], task->regions[1], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 1> acc_bias(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  int batch_size = acc_input.rect.hi[1] - acc_output.rect.lo[1] + 1;
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_bias.rect.volume() == out_dim);

  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, in_dim,
                        &alpha, acc_kernel.ptr, in_dim,
                        acc_input.ptr, in_dim, &beta,
                        acc_output.ptr, out_dim));
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        out_dim, batch_size, 1,
                        &alpha, acc_bias.ptr, 1,
                        m->one_ptr, 1, &alpha,
                        acc_output.ptr, out_dim));
  if (linear->activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, acc_output.ptr,
        &beta, m->outputTensor, acc_output.ptr));
  }
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear forward time = %.2lfms\n", elapsed);
    print_tensor<2, float>(acc_input.ptr, acc_input.rect, "[Linear:forward:input]");
    print_tensor<2, float>(acc_kernel.ptr, acc_kernel.rect, "[Linear:forward:kernel]");
    print_tensor<2, float>(acc_output.ptr, acc_output.rect, "[Linear:forward:output]");
    checkCUDA(hipDeviceSynchronize());
  }
}

void Linear::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(LINEAR_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Linear)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(kernel.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, kernel.region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(bias.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, bias.region));
  launcher.add_field(3, FID_DATA);
  runtime->execute_index_space(ctx, launcher);
}

__global__
void sigmoid_backward(float *grad_ptr, const float *output, int n)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1 - output[i]);
  }
}

/*
  regions[0](I): input
  regions[1](O): replica_grad or input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](O): filter_grad
  regions[6](O): bias_grad
*/
__host__
void Linear::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f, beta = 0.0f;
  const Linear* linear = (Linear*) task->args;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  float* input_grad = NULL;
  TensorAccessorR<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
  TensorAccessorR<float, 2> acc_output(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
  int in_dim = acc_input.rect.hi[0] - acc_input.rect.lo[0] + 1;
  int batch_size = acc_input.rect.hi[1] - acc_input.rect.lo[1] + 1;
  int out_dim = acc_output.rect.hi[0] - acc_output.rect.lo[0] + 1;
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
  if (domain.get_dim() == 3) {
    TensorAccessorW<float, 3> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        false/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  } else {
    TensorAccessorW<float, 2> acc_replica_grad(
        regions[1], task->regions[1], FID_DATA, ctx, runtime,
        false/*readOutput*/);
    assert(acc_replica_grad.rect.volume() == in_dim * batch_size);
    input_grad = acc_replica_grad.ptr;
  }
  TensorAccessorW<float, 2> acc_output_grad(
      regions[3], task->regions[3], FID_DATA, ctx, runtime,
      true/*readOutput*/);
  TensorAccessorR<float, 2> acc_kernel(
      regions[4], task->regions[4], FID_DATA, ctx, runtime);
  TensorAccessorW<float, 2> acc_kernel_grad(
      regions[5], task->regions[5], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorW<float, 1> acc_bias_grad(
      regions[6], task->regions[6], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  // make sure the sizes match
  assert(acc_output.rect.volume() == out_dim * batch_size);
  assert(acc_output_grad.rect.volume() == out_dim * batch_size);
  assert(acc_kernel.rect.volume() == in_dim * out_dim);
  assert(acc_kernel_grad.rect.volume() == in_dim * out_dim);
  assert(acc_bias_grad.rect.volume() == out_dim);
  hipEvent_t t_start, t_end;
  if (linear->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
#ifndef DISABLE_LEGION_CUDA_HIJACK
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
#endif
  if (linear->activation == AC_MODE_RELU) {
    reluBackward<<<GET_BLOCKS(acc_output.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_output_grad.ptr, acc_output.ptr, acc_output.rect.volume());
  } else if (linear->activation == AC_MODE_SIGMOID) {
    sigmoid_backward<<<GET_BLOCKS(acc_output.rect.volume()), CUDA_NUM_THREADS>>>(
        acc_output_grad.ptr, acc_output.ptr, acc_output.rect.volume());
  } else {
    // TODO: only support relu and sigmoid for now
    assert(linear->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        in_dim, out_dim, batch_size,
                        &alpha, acc_input.ptr, in_dim,
                        acc_output_grad.ptr, out_dim,
                        &beta, acc_kernel_grad.ptr, in_dim));
  // Compute bias gradiant
  checkCUDA(hipblasSgemv(m->handle.blas, HIPBLAS_OP_N,
                        out_dim, batch_size,
                        &alpha, acc_output_grad.ptr, out_dim,
                        m->one_ptr, 1,
                        &beta, acc_bias_grad.ptr, 1));
  // Compute data gradiant
  checkCUDA(hipblasSgemm(m->handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        in_dim, batch_size, out_dim,
                        &alpha, acc_kernel.ptr, in_dim,
                        acc_output_grad.ptr, out_dim,
                        &beta, input_grad, in_dim));
  if (linear->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Linear backward time = %.2lfms\n", elapsed);
    print_tensor<2, float>(acc_output_grad.ptr, acc_output_grad.rect, "[Linear:backward:output_grad]");
    print_tensor<2, float>(acc_kernel_grad.ptr, acc_kernel_grad.rect, "[Linear:backward:kernel_grad]");
    print_tensor<1, float>(acc_bias_grad.ptr, acc_bias_grad.rect, "[Linear:backward:bias_grad]");
    print_tensor<2, float>(input_grad, acc_input.rect, "[Linear:backward:input_grad]");
    checkCUDA(hipDeviceSynchronize());
  }
}

/*
  regions[0](O): input_grad
  regions[1](I): replicas
*/
__host__
void Linear::backward2_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  float alpha = 1.0f;
  const LinearMeta* m = *((LinearMeta**) task->local_args);
  TensorAccessorW<float, 2> acc_input(
      regions[0], task->regions[0], FID_DATA, ctx, runtime,
      false/*readOutput*/);
  TensorAccessorR<float, 3> acc_replica(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  assert(acc_input.rect.hi[0] == acc_replica.rect.hi[0]);
  assert(acc_input.rect.lo[0] == acc_replica.rect.lo[0]);
  assert(acc_input.rect.hi[1] == acc_replica.rect.hi[1]);
  assert(acc_input.rect.lo[1] == acc_replica.rect.lo[1]);
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  int num_replica = acc_replica.rect.hi[2] - acc_replica.rect.lo[2] + 1;
  const float *replica_ptr = acc_replica.ptr;
  for (int i = 1; i < num_replica; i++) {
    checkCUDA(hipblasSaxpy(m->handle.blas, acc_input.rect.volume(),
                          &alpha, replica_ptr, 1, acc_input.ptr, 1));
    replica_ptr += acc_input.rect.volume();
  }
}

void Linear::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<2> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<2> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  {
    IndexLauncher launcher(LINEAR_BWD_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    // regions[0](I): input
    launcher.add_region_requirement(
        RegionRequirement(input_lps[0], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[0].region));
    launcher.add_field(0, FID_DATA);
    // regions[1](O): replica_grad 
    if (replica.region_grad != LogicalRegion::NO_REGION) {
      launcher.add_region_requirement(
          RegionRequirement(replica.part_grad, 0/*projection id*/,
                            WRITE_ONLY, EXCLUSIVE, replica.region_grad));
      launcher.add_field(1, FID_DATA);
    } else {
      launcher.add_region_requirement(
          RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                            WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
      launcher.add_field(1, FID_DATA);
    }
    // regions[2](I): output
    launcher.add_region_requirement(
        RegionRequirement(output.part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, output.region));
    launcher.add_field(2, FID_DATA);
    // regions[3](I/O): output_grad
    launcher.add_region_requirement(
        RegionRequirement(output.part_grad, 0/*projection id*/,
                          READ_WRITE, EXCLUSIVE, output.region_grad));
    launcher.add_field(3, FID_DATA);
    // regions[4](I): filter
    launcher.add_region_requirement(
        RegionRequirement(kernel.part, 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, kernel.region));
    launcher.add_field(4, FID_DATA);
    // regions[5](O): filter_grad
    launcher.add_region_requirement(
        RegionRequirement(kernel.part_grad, 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, kernel.region_grad));
    launcher.add_field(5, FID_DATA);
    // regions[6](O): bias_grad
    launcher.add_region_requirement(
        RegionRequirement(bias.part_grad, 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, bias.region_grad));
    launcher.add_field(6, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
  if (replica.region_grad != LogicalRegion::NO_REGION) {
    // We aggregate parameters from replica tensor to input tensor
    // Note we use input's task_is to reduce extra data transfers
    Rect<2> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part_grad.get_index_partition());
    IndexSpaceT<2> input_task_is = IndexSpaceT<2>(ff.get_task_is(input_rect));
    IndexLauncher launcher(LINEAR_BWD2_TASK_ID, task_is,
                           TaskArgument(this, sizeof(Linear)), argmap,
                           Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                           FFConfig::get_hash_id(std::string(name)));
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[0], 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
    launcher.add_field(0, FID_DATA);
    // Note that replica.part save's a partition of replica.region_grad
    launcher.add_region_requirement(
        RegionRequirement(replica.part, 0/*partition id*/,
                          READ_ONLY, EXCLUSIVE, replica.region_grad));
    launcher.add_field(1, FID_DATA);
    runtime->execute_index_space(ctx, launcher);
  }
}

