#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::concat(std::string name,
                       int n, const Tensor* tensors,
                       int axis)
{
  Concat *cat = new Concat(*this, name, n, tensors, axis);
  layers.push_back(cat);
  return cat->output;
}

Concat::Concat(FFModel& model,
               const std::string& pcname, 
               int _n, const Tensor* _tensors,
               int _axis)
 : Op(pcname, _n, _tensors), axis(_axis),
   profiling(model.config.profiling)
{
  // Retrive the task indexspace for the op
  task_is = model.get_or_create_task_is(pcname);

  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  FieldSpace fs = model.config.field_space;
  int dims[MAX_DIM], num_dim = inputs[0].numDim;
  assert(num_dim == domain.get_dim());
  for (int i = 0; i < num_dim; i++)
    dims[i] = inputs[0].adim[num_dim-1-i];
  for (int i = 1; i < numInputs; i++)
    for (int j = 0; j < num_dim; j++) {
      if (j != axis)
        assert(inputs[i].adim[num_dim-1-j] == dims[j]);
      else
        dims[j] += inputs[i].adim[num_dim-1-j];
    }
  for (int i = 0; i < num_dim; i++)
    printf("concat: dim[%d] = %d\n", i, dims[i]);
  switch (domain.get_dim()) {
    case 1:
    {
      Rect<1> part_rect = domain;
      output = model.create_tensor<1>(dims, IndexSpaceT<1>(task_is), DT_FLOAT);
      for (int i = 0; i < numInputs; i++) {
        Rect<1> input_rect = runtime->get_index_partition_color_space(
            ctx, inputs[i].part.get_index_partition());
        if (input_rect == part_rect) {
          input_lps[i] = inputs[i].part;
          input_grad_lps[i] = inputs[i].part_grad;
        } else {
          model.create_disjoint_partition<1>(inputs[i],
              IndexSpaceT<1>(task_is), input_lps[i], input_grad_lps[i]);
        }
      }
      break;
    }
    case 2:
    {
      Rect<2> part_rect = domain;
      output = model.create_tensor<2>(dims, IndexSpaceT<2>(task_is), DT_FLOAT);
      for (int i = 0; i < numInputs; i++) {
        Rect<2> input_rect = runtime->get_index_partition_color_space(
            ctx, inputs[i].part.get_index_partition());
        if (input_rect == part_rect) {
          input_lps[i] = inputs[i].part;
          input_grad_lps[i] = inputs[i].part_grad;
        } else {
           model.create_disjoint_partition<2>(inputs[i],
               IndexSpaceT<2>(task_is), input_lps[i], input_grad_lps[i]);
        }
      }
      break;
    }
    default:
    {
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
    }
  }
#ifdef DEADCODE
  int num_par_w = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_h = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_par_n = part_rect.hi[2] - part_rect.lo[2] + 1;
  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int input_c = 0;
  int input_n = inputs[0].adim[3];
  for (int i = 0; i < numInputs; i++) {
    assert(input_w == inputs[i].adim[0]);
    assert(input_h == inputs[i].adim[1]);
    assert(input_n == inputs[i].adim[3]);
    input_c += inputs[i].adim[2];
  }
  int input_nc = input_n * input_c;
  Rect<3, coord_t> output_rect(Point<3>(0, 0, 0),
                      Point<3>(input_w-1, input_h-1, input_nc-1));
  IndexSpaceT<3> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr = runtime->create_logical_region(ctx, output_is, fs);
  Transform<3, 3, coord_t> transform;
  int extent_w = (input_w + num_par_w - 1) / num_par_w;
  int extent_h = (input_h + num_par_h - 1) / num_par_h;
  int extent_nc = input_nc / num_par_n;
  assert(input_nc % num_par_n == 0);
  Rect<3, coord_t> extent(Point<3>(0, 0, 0), Point<3>(extent_w-1, extent_h-1, extent_nc-1));
  transform[0][0] = extent_w; transform[0][1] = 0; transform[0][2] = 0;
  transform[1][0] = 0; transform[1][1] = extent_h; transform[1][2] = 0;
  transform[2][0] = 0; transform[2][1] = 0; transform[2][2] = extent_nc;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, task_is, transform, extent);
  assert(runtime->is_index_partition_disjoint(ctx, output_ip));
  assert(runtime->is_index_partition_complete(ctx, output_ip));
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp =
    runtime->get_logical_partition(ctx, output_grad_lr, output_ip);

  output.numDim = 4;
  output.adim[0] = input_w;
  output.adim[1] = input_h;
  output.adim[2] = input_c;
  output.adim[3] = inputs[0].adim[3];
  output.pdim[0] = extent_w;
  output.pdim[1] = extent_h;
  output.pdim[2] = input_c;
  output.pdim[3] = extent_nc / input_c;
  assert(extent_nc % input_c == 0);
  output.region = output_lr;
  output.part = output_lp;
  output.region_grad = output_grad_lr;
  output.part_grad = output_grad_lp;
  printf("Create concat layer: output(n=%d c=%d h=%d w=%d)\n",
         output.adim[3], output.adim[2], output.adim[1], output.adim[0]);
  for (int i = 0; i < numInputs; i++) {
    // For now, we assume our output has the same partition as all inputs
    Rect<3> input_part_rect =
      runtime->get_index_partition_color_space(ctx, inputs[i].part.get_index_partition());
    assert(part_rect == input_part_rect);
    input_lps[i] = inputs[i].part;
  }
#endif
}

#ifdef DEADCODE
__host__
OpMeta* Concat::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  FFHandler handler = *((const FFHandler*) task->local_args);
  ConcatMeta* m = new ConcatMeta(handler);
  return m;
}
#endif

void Concat::init(const FFModel& ff)
{
#ifdef DEADCODE
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    FFHandler handler = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handler, sizeof(FFHandler)));
  }
  IndexLauncher init_launcher(CONCAT_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(Concat)), argmap);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
#endif
}

__global__
void add_with_stride(float* output,
                     const float* input,
                     int num_blocks,
                     int output_blk_size,
                     int input_blk_size)
{
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size)
  {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] += input[input_offset];
  }
}

/*
  regions[0](O): output
  regions[1..numInputs](I): inputs
*/
void Concat::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  int axis = cc->output.numDim - 1 - cc->axis;
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  float *output;
  const float *inputs[MAX_NUM_INPUTS];
  int num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  for (int d = 0; d < cc->output.numDim; d++) {
    if (d <= axis)
      output_blk_size *= cc->output.adim[d];
    else
      num_blocks *= cc->output.adim[d];
  }
  for (int i = 0; i < cc->numInputs; i++) {
    input_blk_sizes[i] = 1;
    for (int d = 0; d <= axis; d++)
      input_blk_sizes[i] *= cc->inputs[i].adim[d];
  }
  assert(cc->numInputs <= MAX_NUM_INPUTS);
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  assert(domain.get_dim() == cc->output.numDim);
  switch (domain.get_dim()) {
    case 1:
    {
      TensorAccessorW<float, 1> accOutput(
          regions[0], task->regions[0], FID_DATA, ctx, runtime,
          false/*readOutput*/);
      output = accOutput.ptr;
      for (int i = 0; i < cc->numInputs; i++) {
        TensorAccessorR<float, 1> accInput(
            regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);
        inputs[i] = accInput.ptr;
      }
      break;
    }
    case 2:
    {
      TensorAccessorW<float, 2> accOutput(
          regions[0], task->regions[0], FID_DATA, ctx, runtime,
          false/*readOutput*/);
      output = accOutput.ptr;
      for (int i = 0; i < cc->numInputs; i++) {
        TensorAccessorR<float, 2> accInput(
            regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime);
        inputs[i] = accInput.ptr;
      }
      break;
    }
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }
  for (int i = 0; i < cc->numInputs; i++) {
    add_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS>>>(
        output, inputs[i], num_blocks, output_blk_size, input_blk_sizes[i]);
    output += input_blk_sizes[i];
  }
  checkCUDA(hipDeviceSynchronize());
  if (cc->profiling) {
    Rect<2> rect(Point<2>(0, 0), Point<2>(output_blk_size-1, domain.get_volume() / output_blk_size - 1));
    print_tensor<2, float>(output - output_blk_size, rect, "[Concat:forward:output]");
  }
#ifdef DEADCODE
  const AccessorWO<float, 3> acc_output(regions[0], FID_DATA);
  Rect<3> rect_output;
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  float *output_ptr = acc_output.ptr(rect_output.lo);
  float *output_bound = output_ptr + rect_output.volume();
  for (int i = 0; i < cc->numInputs; i++) {
    const AccessorRO<float, 3> acc_input(regions[i+1], FID_DATA);
    Rect<3> rect_input =
      runtime->get_index_space_domain(ctx, task->regions[i+1].region.get_index_space());
    assert(acc_input.accessor.is_dense_arbitrary(rect_input));
    const float *input_ptr = acc_input.ptr(rect_input.lo);
    checkCUDA(hipMemcpyAsync(output_ptr, input_ptr,
                              rect_input.volume() * sizeof(float),
                              hipMemcpyDeviceToDevice));
    output_ptr += rect_input.volume();
  }
  assert(output_ptr == output_bound);
#endif
}

void Concat::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
#ifdef DEADCODE
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
#endif
  IndexLauncher launcher(CONCAT_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Concat)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_ONLY, EXCLUSIVE, output.region,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
        RegionRequirement(input_lps[i], 0/*projection id*/,
                          READ_ONLY, EXCLUSIVE, inputs[i].region,
                          MAP_TO_ZC_MEMORY));
    launcher.add_field(i + 1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I): output_grad
  regions[1..numInputs](O): input_grad
*/
void Concat::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  const Concat* cc = (Concat*) task->args;
  // Note that our internal axis index ordering is opposite to other frameworks
  int axis = cc->output.numDim - 1 - cc->axis;
  assert(regions.size() == cc->numInputs + 1);
  assert(task->regions.size() == cc->numInputs + 1);
  const float *output_grad;
  float *input_grads[MAX_NUM_INPUTS];
  int num_blocks = 1, output_blk_size = 1, input_blk_sizes[MAX_NUM_INPUTS];
  for (int d = 0; d < cc->output.numDim; d++) {
    if (d <= axis)
      output_blk_size *= cc->output.adim[d];
    else
      num_blocks *= cc->output.adim[d];
  }
  for (int i = 0; i < cc->numInputs; i++) {
    input_blk_sizes[i] = 1;
    for (int d = 0; d <= axis; d++)
      input_blk_sizes[i] *= cc->inputs[i].adim[d];
  }
  assert(cc->numInputs <= MAX_NUM_INPUTS);
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  assert(domain.get_dim() == cc->output.numDim);
  switch (domain.get_dim()) {
    case 1:
    {
      TensorAccessorR<float, 1> accOutputGrad(
          regions[0], task->regions[0], FID_DATA, ctx, runtime);
      output_grad = accOutputGrad.ptr;
      for (int i = 0; i < cc->numInputs; i++) {
        TensorAccessorW<float, 1> accInputGrad(
            regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime,
            false/*readOutput*/);
        input_grads[i] = accInputGrad.ptr;
      }
      break;
    }
    case 2:
    {
      TensorAccessorR<float, 2> accOutputGrad(
          regions[0], task->regions[0], FID_DATA, ctx, runtime);
      output_grad = accOutputGrad.ptr;
      for (int i = 0; i < cc->numInputs; i++) {
        TensorAccessorW<float, 2> accInputGrad(
            regions[i+1], task->regions[i+1], FID_DATA, ctx, runtime,
            false/*readOutput*/);
        input_grads[i] = accInputGrad.ptr;
      }
      break;
    }
    default:
      fprintf(stderr, "Unsupported concat dimension number");
      assert(false);
  }
  for (int i = 0; i < cc->numInputs; i++) {
    add_with_stride<<<GET_BLOCKS(input_blk_sizes[i]*num_blocks), CUDA_NUM_THREADS>>>(
        input_grads[i], output_grad, num_blocks, input_blk_sizes[i], output_blk_size);
    output_grad += input_blk_sizes[i];
  }
  checkCUDA(hipDeviceSynchronize());
  if (cc->profiling) {
    int batch_size = domain.get_volume() / output_blk_size;
    Rect<2> output_rect(Point<2>(0, 0), Point<2>(output_blk_size-1, batch_size - 1));
    Rect<2> input_rect(Point<2>(0, 0), Point<2>(input_blk_sizes[0]-1, batch_size - 1));
    print_tensor<2, float>(output_grad - output_blk_size, output_rect, "[Concat:forward:output]");
    print_tensor<2, float>(input_grads[0], input_rect, "[Concat:forward:input0]");
  }
#ifdef DEADCODE
  const AccessorRO<float, 3> acc_output(regions[0], FID_DATA);
  Rect<3> rect_output;
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  float *output_ptr = (float*) acc_output.ptr(rect_output.lo);
  float *output_bound = output_ptr + rect_output.volume();
  for (int i = 0; i < cc->numInputs; i++) {
    const AccessorWO<float, 3> acc_input(regions[i+1], FID_DATA);
    Rect<3> rect_input =
      runtime->get_index_space_domain(ctx, task->regions[i+1].region.get_index_space());
    assert(acc_input.accessor.is_dense_arbitrary(rect_input));
    float *input_ptr = acc_input.ptr(rect_input.lo);
    checkCUDA(hipMemcpyAsync(input_ptr, output_ptr,
                              rect_input.volume() * sizeof(float),
                              hipMemcpyDeviceToDevice));
    output_ptr += rect_input.volume();
  }
  assert(output_ptr == output_bound);
#endif
}

void Concat::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
#ifdef DEADCODE
  Rect<3> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<3> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
#endif
  IndexLauncher launcher(CONCAT_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Concat)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, output.region_grad,
                        MAP_TO_ZC_MEMORY));
  launcher.add_field(0, FID_DATA);
  for (int i = 0; i < numInputs; i++) {
    launcher.add_region_requirement(
        RegionRequirement(input_grad_lps[i], 0/*projection id*/,
                          WRITE_ONLY, EXCLUSIVE, inputs[i].region_grad,
                          MAP_TO_ZC_MEMORY));
    launcher.add_field(i + 1, FID_DATA);
  }
  runtime->execute_index_space(ctx, launcher);
}

//void Concat::update(const FFModel& ff)
//{
//}
